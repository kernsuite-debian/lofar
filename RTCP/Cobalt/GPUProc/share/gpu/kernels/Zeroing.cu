#include "hip/hip_runtime.h"
//# Zeroing.cu: zero ranges of samples
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include "gpu_math.cuh"
#include <stdio.h>

typedef float2 FilteredDataType[NR_STABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS];

typedef char MaskType[NR_STABS][NR_SAMPLES_PER_CHANNEL];

/**
 * Zero samples that have been flagged. Clears samples for all channels for
 * ranged specified per station.
 *
 * @param[data] a multi-dimensional array with time samples of type complex
 * float in the last dimension.
 * @param[mask] an 2D array of bytes, each representing a sample of a station.
 * A value of 0 means ignore this sample, a value of 1 means zero this sample.
 */

extern "C"
{
  __global__ void Zeroing(FilteredDataType data,
                           MaskType mask)
  {
    int sample  = blockIdx.x * blockDim.x + threadIdx.x;
    int channel = (blockIdx.y * blockDim.y + threadIdx.y);
    int station = (blockIdx.z * blockDim.z + threadIdx.z) / 2;
    int pol     = (blockIdx.z * blockDim.z + threadIdx.z) % 2;

    if (mask[station][sample]) {
      // Clear our sample
      data[station][pol][sample][channel] = make_float2(0.0f, 0.0f);
    }
  }
}
