#include "hip/hip_runtime.h"
//# IntToFloat.cu: Convert integer input to float; transpose time and pol dims
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include "IntToFloat.cuh"

// SampleType is a COMPLEX sample
#if NR_BITS_PER_SAMPLE ==  4
typedef signed char SampleType;
#elif NR_BITS_PER_SAMPLE ==  8
typedef char2  SampleType;
#elif NR_BITS_PER_SAMPLE == 16
typedef short2 SampleType;
#else
#error unsupported NR_BITS_PER_SAMPLE: must be 4, 8, or 16
#endif

#if NR_BITS_PER_SAMPLE ==  4
#define REAL(sample) extractRI(sample, false)
#define IMAG(sample) extractRI(sample, true)
#else
#define REAL(sample) sample.x
#define IMAG(sample) sample.y
#endif

typedef SampleType (*SampledDataType)  [NR_INPUT_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
typedef float2     (*ConvertedDataType)[NR_OUTPUT_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_SUBBAND];

/**
 * This kernel performs a conversion of the integer valued input to floats and
 * transposes the data to get per station: first all samples with polX, then polY.
 * - It supports 8 and 16 bit (char and short) input, which is selectable using
 *   the define NR_BITS_PER_SAMPLE
 * - In 8 bit mode the converted samples with value -128 are clamped to -127.0f
 *
 * @param[out] convertedDataPtr    pointer to output data of ConvertedDataType,
 *                                 a 4D array [station_out][polarizations][n_samples_subband][complex]
 *                                 of floats (2 complex polarizations).
 * @param[in]  sampledDataPtr      pointer to input data; this can either be a
 *                                 4D array [station_in][n_samples_subband][polarizations][complex]
 *                                 of shorts or chars, depending on NR_BITS_PER_SAMPLE.
 * @param[in]  stationIndices      list of input stations to use for each output station
 *                                 1D array [station_out]
 *                                 of unsigned
 *
 * Required preprocessor symbols:
 * - NR_SAMPLES_PER_CHANNEL: > 0
 * - NR_BITS_PER_SAMPLE: 8 or 16
 * - NR_INPUT_STATIONS
 * - NR_OUTPUT_STATIONS
 *
 * Optional preprocessor symbols:
 * - DO_FFTSHIFT, if an fft-shift is to be performed as well
 * - DO_STATIONSUBSET, if the stationIndices input array is to be used to select a subset
 *                     of stations.
 *
 * Execution configuration:
 * - Use a 1D thread block. No restrictions.
 * - Use a 2D grid dim, where the x dim has 1 block and the y dim represents the
 *   number of output stations (i.e. antenna fields).
 */

extern "C" {
__global__ void intToFloat(void *convertedDataPtr,
                           const void *sampledDataPtr,
                           const unsigned *stationIndices)
{
  ConvertedDataType convertedData = (ConvertedDataType)convertedDataPtr;
  SampledDataType   sampledData   = (SampledDataType)  sampledDataPtr;

#ifdef DO_STATIONSUBSET
  uint station_in  = stationIndices[blockIdx.y];
  uint station_out = blockIdx.y;
#else
  uint station_in  = blockIdx.y;
  uint station_out = blockIdx.y;
#endif

#ifdef DO_FFTSHIFT
  // Multiplication factor: 1 for even samples, -1 for odd samples
  //const float factor = 1 - 2 * (threadIdx.x % 2); 
  const float factor = threadIdx.x % 2 ? -1.0f : +1.0f;
#else
  const float factor = 1.0f;
#endif

  // For even increases, we always process either even or odd samples
  for (int time = threadIdx.x; time < NR_SAMPLES_PER_SUBBAND; time += blockDim.x)
  {
    float4 sample;

    sample = make_float4(convertIntToFloat(REAL((*sampledData)[station_in][time][0])) * factor,
                         convertIntToFloat(IMAG((*sampledData)[station_in][time][0])) * factor,
                         convertIntToFloat(REAL((*sampledData)[station_in][time][1])) * factor,
                         convertIntToFloat(IMAG((*sampledData)[station_in][time][1])) * factor);

    float2 sampleX = make_float2(sample.x, sample.y);
    (*convertedData)[station_out][0][time] = sampleX;
    float2 sampleY = make_float2(sample.z, sample.w);
    (*convertedData)[station_out][1][time] = sampleY;
  }

}

}

